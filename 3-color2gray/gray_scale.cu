#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <stdexcept>
#include <string>

#include "gray_scale.h"

__global__ void BGRToGrayscaleKernel(uchar* Pout, uchar* Pin, int width, int height) {
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int p = row*width + col; // (flattened) position of the pixel
        int i = p*3;
        uchar b = Pin[i];
        uchar g = Pin[i+1];
        uchar r = Pin[i+2];
        Pout[p] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

static void linearize3D(uchar* out, uchar*** arr, int x_dim, int y_dim, int z_dim) {

    int index = 0;
    for (int x = 0; x < x_dim; ++x) {
        for (int y = 0; y < y_dim; ++y) {
            for (int z = 0; z < z_dim; ++z) {
                out[index++] = arr[x][y][z];
            }
        }
    }
}

static void reshape2D(uchar** out, uchar* vec, int x_dim, int y_dim) {
    for (int x = 0; x < x_dim; ++x) {
        for (int y = 0; y < y_dim; ++y) {
            int i = (x*x_dim)+y;
            out[x][y] = vec[i];
        }
    }
}

void BGRToGrayscale(uchar** out, uchar*** image, int width, int height) {
    int n_pixel = width * height;
    int input_size = n_pixel * CHANNEL_NUMBER * sizeof(uchar);
    int output_size = n_pixel * sizeof(uchar);

    // This is used both for host-side input and output.
    assert(input_size >= output_size);
    uchar* P_h = new uchar[input_size];
    linearize3D(P_h, image, height, width, CHANNEL_NUMBER);

    uchar* Pin_d, *Pout_d;
    hipMalloc((void **)&Pin_d, input_size);
    hipMalloc((void **)&Pout_d, output_size);

    hipMemcpy(Pin_d, P_h, input_size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(ceil(height/16.0), ceil(width/16.0), 1);
    BGRToGrayscaleKernel<<<dimGrid, dimBlock>>>(Pout_d, Pin_d, width, height);

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        std::string str{};
        str += "Kernel launch error: ";
        str += hipGetErrorString(err);
        throw std::runtime_error{str};
    }

    hipMemcpy(P_h, Pout_d, output_size, hipMemcpyDeviceToHost);
    reshape2D(out, P_h, height, width);

    hipFree(Pin_d);
    hipFree(Pout_d);

    delete []P_h;
}
